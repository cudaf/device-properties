#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include "error.h"


// The kernel recieves 3 arguments, the first being global address (GPU) of
// where it must store the result. This has to be done because the kernel cant
// return any value. The arguments it recieves are managed by CUDA driver and
// possibly stored in constant memory (right?). A kernel supports all common
// operators along with various math functions.
__global__ void kernel(int *c, int a, int b) {
  *c = a + b;
}


// 1. Integers "a", "b" are defined in host memory (CPU).
// 2. Memory for storing their sum is allocated on device memory (GPU).
// 3. Sum is computed by the kernel, with one thread (async).
// 4. Wait for kernel to complete, then copy the sum to host memory (cHost).
// 5. Free the space we had occupied (we are good people).
int main() {
  int a = 1, b = 2; // 1
  int cHost, *cDevice;                      // 2
  TRY( hipMalloc(&cDevice, sizeof(int)) ); // 2
  kernel<<<1, 1>>>(cDevice, a, b); // 3
  TRY( hipMemcpy(&cHost, cDevice, sizeof(int), hipMemcpyDeviceToHost) ); // 4
  TRY( hipFree(cDevice) ); // 5
  printf("a = %d, b = %d\n", a, b);
  printf("a + b = %d (GPU)\n", cHost);
  return 0;
}
