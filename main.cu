#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include "error.h"


// 1. Check how many compute devices are attached.
// 2. List some properties of each device.
int main() {
  int N;                         // 1
  TRY( hipGetDeviceCount(&N) ); // 1

  hipDeviceProp_t p;                        // 2
  for (int i=0; i<N; i++) {                // 2
    TRY( hipGetDeviceProperties(&p, i) ); // 2
    printf("COMPUTE DEVICE %d:\n", i);
    printf("Name: %s\n", p.name);
    printf("Compute capability: %d.%d\n", p.major, p.minor);
    printf("Multiprocessors: %d\n", p.multiProcessorCount);
    printf("Clock rate: %d MHz\n", p.clockRate / 1000);
    printf("Global memory: %lld MB\n", p.totalGlobalMem / (1024*1024));
    printf("Constant memory: %lld KB\n", p.totalConstMem / 1024);
    printf("Shared memory per block: %lld KB\n", p.sharedMemPerBlock / 1024);
    printf("Registers per block: %d\n", p.regsPerBlock);
    printf("Threads per block: %d (max)\n", p.maxThreadsPerBlock);
    printf("Threads per warp: %d\n", p.warpSize);
    printf("Block dimension: %dx%dx%d (max)\n", p.maxThreadsDim[0], p.maxThreadsDim[1], p.maxThreadsDim[2]);
    printf("Grid dimension: %dx%dx%d (max)\n", p.maxGridSize[0], p.maxGridSize[1], p.maxGridSize[2]);
    printf("Device copy overlap: %s\n", p.deviceOverlap ? "yes" : "no");
    printf("Kernel execution timeout: %s\n", p.kernelExecTimeoutEnabled ? "yes" : "no");
    printf("\n");
  }
  return 0;
}
